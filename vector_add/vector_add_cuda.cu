
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define N 1000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    auto start = std::chrono::system_clock::now();
    float *a, *b, *out; 
    float *d_a, *d_b, *d_out; 

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f * (i % 10); 
        b[i] = 2.0f * (i % 10);
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Main function
    vector_add<<<1,1>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Cleanup after kernel execution
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    auto end = std::chrono::system_clock::now();

    std::chrono::duration<double> elapsed_seconds = end-start;
 
    std::cout << "elapsed time: " << elapsed_seconds.count() << "s"
              << std::endl;

    std::cout << "a: ";
    for (int i = 0; i < 10; ++i) std::cout << a[i] << ",";
    std::cout << std::endl;

    std::cout << "b: ";
    for (int i = 0; i < 10; ++i) std::cout << b[i] << ",";
    std::cout << std::endl;
    
    std::cout << "out: ";
    for (int i = 0; i < 10; ++i) std::cout << out[i] << ",";
    std::cout << std::endl;

    free(a);
    free(b);
    free(out);
}
