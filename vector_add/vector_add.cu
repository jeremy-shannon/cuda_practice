
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define N 1000000000

void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    auto start = std::chrono::system_clock::now();
    float *a, *b, *out; 

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Main function
    vector_add(out, a, b, N);

    auto end = std::chrono::system_clock::now();

    std::chrono::duration<double> elapsed_seconds = end-start;
 
    std::cout << "elapsed time: " << elapsed_seconds.count() << "s"
              << std::endl;

    free(a);
    free(b);
    free(out);
}
