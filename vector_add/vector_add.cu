
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define N 1000000000

void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    auto start = std::chrono::system_clock::now();
    float *a, *b, *out; 

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f * (i % 10); 
        b[i] = 2.0f * (i % 10);
    }

    // Main function
    vector_add(out, a, b, N);

    auto end = std::chrono::system_clock::now();

    std::chrono::duration<double> elapsed_seconds = end-start;
 
    std::cout << "elapsed time: " << elapsed_seconds.count() << "s"
              << std::endl;

    std::cout << "a: ";
    for (int i = 0; i < 10; ++i) std::cout << a[i] << ",";
    std::cout << std::endl;

    std::cout << "b: ";
    for (int i = 0; i < 10; ++i) std::cout << b[i] << ",";
    std::cout << std::endl;
    
    std::cout << "out: ";
    for (int i = 0; i < 10; ++i) std::cout << out[i] << ",";
    std::cout << std::endl;

    free(a);
    free(b);
    free(out);
}
