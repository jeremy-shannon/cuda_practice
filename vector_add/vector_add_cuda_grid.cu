
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define N 100000000

using namespace std;

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Handling arbitrary vector size
    if (tid < n){
        out[tid] = a[tid] + b[tid];
    }
}

int main(){
    auto start = chrono::system_clock::now();
    float *a, *b, *out; 
    float *d_a, *d_b, *d_out; 

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f * (i % 10); 
        b[i] = 2.0f * (i % 10);
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Executing kernel 
    int block_size = 16*16 * 8;
    int grid_size = ((N + block_size) / block_size);

    cout << "block size: " << block_size << ", grid size: " << grid_size << endl;
    
    vector_add<<<grid_size,block_size>>>(d_out, d_a, d_b, N);
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) { 
        cout << hipGetErrorString(error) << endl;
    }

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Cleanup after kernel execution
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    auto end = chrono::system_clock::now();

    chrono::duration<double> elapsed_seconds = end-start;
 
    cout << "elapsed time: " << elapsed_seconds.count() << "s"
              << endl;

    cout << "a: ";
    for (int i = 0; i < 10; ++i) cout << a[i] << ",";
    cout << endl;

    cout << "b: ";
    for (int i = 0; i < 10; ++i) cout << b[i] << ",";
    cout << endl;
    
    cout << "out: ";
    for (int i = 0; i < 10; ++i) cout << out[i] << ",";
    cout << endl;

    free(a);
    free(b);
    free(out);
}
