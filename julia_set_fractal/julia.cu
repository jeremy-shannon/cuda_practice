#include "hip/hip_runtime.h"
#include "cpu_bitmap.h"
#include "julia.cuh"
#include <memory>
 
using namespace std;

struct hipComplex {
    float r;
    float i;
    __device__ hipComplex( float a, float b ) : r(a), i(b) {}
    __device__ float magnitude2( void ) {
        return r * r + i * i;
    }
    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
};

__device__ int julia( int x, int y, int w, int h) {
	const float scale = 1.5;
	float jx = scale * (float)(w/2 - x)/(w/2);
	float jy = scale * (float)(h/2 - y)/(h/2);
	
	hipComplex c(-0.8, 0.156);
	hipComplex a(jx, jy);

	int i = 0;
	for(i=0; i<200; i++){
		a = a * a + c;
		if (a.magnitude2() > 1000)
			return 0;
	}

	return 1;
}

__global__ void kernel( unsigned char *ptr, int w, int h ) {
	// map from threadIdx/BlockIdx to pixel position
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;

	// now calculate the value at that position
	int juliaValue = julia( x, y, w, h);
	ptr[offset*4 + 0] = 255 * juliaValue * 0.8;
	ptr[offset*4 + 1] = 255 * juliaValue * 0.7;
	ptr[offset*4 + 2] = 255 * juliaValue * 0.9;
	ptr[offset*4 + 3] = 255;
}

CPUBitmap* generate_julia(int width, int height ) {
	CPUBitmap* bitmap = new CPUBitmap( width, height );
	unsigned char *dev_bitmap;

	hipMalloc( (void**)&dev_bitmap, bitmap->image_size() ) ;

	dim3 grid( width, height );
	
	kernel<<<grid,1>>>( dev_bitmap, width, height );

	hipMemcpy( bitmap->get_ptr(), 
				  dev_bitmap, 
				  bitmap->image_size(), 
				  hipMemcpyDeviceToHost ) ;

	hipFree( dev_bitmap ) ;

	return bitmap;
}